﻿#include <hip/hip_runtime.h>
#include <stdio.h>

// Helper function to convert compute capability to the number of cores
int ConvertSMVer2Cores(int major, int minor) {
    // Refer to NVIDIA CUDA Programming Guide for the compute capability to cores conversion
    // This is a simplified version and may not cover all cases
    int cores;

    switch ((major << 4) + minor) {
    case 0x10:
        cores = 8;
        break;
    case 0x11:
    case 0x12:
        cores = 8;
        break;
    case 0x13:
        cores = 32;
        break;
    case 0x20:
        cores = 32;
        break;
    default:
        cores = 0;
        break;
    }

    return cores;
}


int main() {
    // CUDA initialization code (if needed)
    hipSetDevice(0); // Set the device to GPU 0 (or the appropriate GPU index)

    int num_devices;
    hipGetDeviceCount(&num_devices);

    if (num_devices == 0) {
        fprintf(stderr, "No CUDA devices found.\n");
        return 1;
    }

    for (int device_id = 0; device_id < num_devices; ++device_id) {
        hipDeviceProp_t device_prop;
        hipError_t cuda_status = hipGetDeviceProperties(&device_prop, device_id);

        if (cuda_status != hipSuccess) {
            fprintf(stderr, "Error: hipGetDeviceProperties failed with error code %d\n", cuda_status);
            return 1;
        }

        printf("Device %d Information:\n", device_id);
        printf("Name: %s\n", device_prop.name);
        printf("Compute Capability: %d.%d\n", device_prop.major, device_prop.minor);
        printf("Clock Rate: %d kHz\n", device_prop.clockRate);
        printf("Number of SM (Streaming Multiprocessors): %d\n", device_prop.multiProcessorCount);
        printf("Number of Cores per SM: %d\n", ConvertSMVer2Cores(device_prop.major, device_prop.minor) * device_prop.multiProcessorCount);
        printf("Warp Size: %d\n", device_prop.warpSize);
        printf("Global Memory Size: %zu bytes\n", device_prop.totalGlobalMem);
        printf("Constant Memory Size: %zu bytes\n", device_prop.totalConstMem);
        printf("Shared Memory Size per Block: %zu bytes\n", device_prop.sharedMemPerBlock);
        printf("Registers per Block: %d\n", device_prop.regsPerBlock);
        printf("Max Threads per Block: %d\n", device_prop.maxThreadsPerBlock);
        printf("Max Size of Each Dimension of a Block: (%d, %d, %d)\n",
            device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1], device_prop.maxThreadsDim[2]);
        printf("Max Size of Each Dimension of a Grid: (%d, %d, %d)\n",
            device_prop.maxGridSize[0], device_prop.maxGridSize[1], device_prop.maxGridSize[2]);

        // Additional analysis or discussions can be included here

        printf("\n");
    }

    // Other CUDA-related code

    return 0;
}

