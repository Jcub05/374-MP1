#include "hip/hip_runtime.h"
﻿// Jacob Badali 20290739
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define Threads_per_block 128

#define num_row (100) //Not sure what this size means, will find out later. wondering why it isn't like [][]...
#define num_col (100)

int  dM[num_row][num_col];
int  dN[num_row][num_col];
int  dP[num_row][num_col];

int  hM[num_row][num_col];
int  hN[num_row][num_col];
int  hP[num_row][num_col];

#define WIDTH = 100; //change this

int size = WIDTH * WIDTH * sizeof(float);



hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


//Multiplication kernel function
__global__ void mulKernel(int* P, int* M, int* N, int rows_and_cols) {
    int rows = blockIdx.y * blockDim.y + threadIdx.y;
    int cols = blockIdx.x * blockDim.x + threadIdx.x;

    float temp_sum = 0.0;

    if (row < rows_and_cols && cols < rows_and_cols) {
        
        for (int i = 0; i < rows_and_cols; i++) {
            tmpSum += P[rows * rows_and_cols + i] * M[i * rows_and_cols + cols];
        }
    }
    C[rows * rows_and_cols + cols] = temp_sum;
}

void matrixMultiplicationdefn(int* P, int* M, int* N, int rows_and_cols) {
    //Define the dimensions of blocks and grids
    dim3 blocksPerGrid(rows_and_cols); //1x1x1
    dim3 threadsPerBlock(1, 1); //100x1x1
}

int main()
{
    //Pointers to host and device memory
    int* h_pointer = 0;
    int* d_pointer = 0;

    int n =






        int matrix_size = 100;
    //fill host mem
    for (int i = 0; i < matrix_size; i++) {
        hN[i] = hM[i] = (float)sqrtf((float)i);
    }

    hipMemcpyAsync(d_pointer, h_pointer, )
        kernel << <blocksPerGrid, threadsPerBlock, 0, 0 >> > ()

}
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
