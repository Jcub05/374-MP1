#include "hip/hip_runtime.h"
﻿// Jacob Badali 20290739
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define Threads_per_block 128

#define num_row (100) //Not sure what this size means, will find out later. wondering why it isn't like [][]...
#define num_col (100)

float  d_M[num_row][num_col];
float  d_N[num_row][num_col];
float  d_P[num_row][num_col];

float  h_M[num_row][num_col];
float  h_N[num_row][num_col];
float  h_P[num_row][num_col];

#define WIDTH = 100; //change this
#define BLOCK_WIDTH = 32;

int size = WIDTH * WIDTH * sizeof(float);



hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


//Multiplication kernel function
__global__ void mulKernel(int* M, int* N, int* P, int size) {
    int rows = blockIdx.y * blockDim.y + threadIdx.y;
    int cols = blockIdx.x * blockDim.x + threadIdx.x;


    if (rows < size && cols < size) {
        float temp_sum = 0.0;
        for (int i = 0; i < size; i++) {
            temp_sum += M[rows * size + i] * N[i * size + cols];
        }
        C[rows * size + cols] = temp_sum;
    }
}


int main()
{
    //Pointers to host and device memory
    int* h_pointer = 0;
    int* d_pointer = 0;

    int NumBlocks = WIDTH / BLOCK_WIDTH;
    if (WIDTH % BLOCK_WIDTH) NumBlocks++;

    dim3 dimGrid(NumBlocks, NumBlocks);
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);

    //Allocate appropriate memory size for each array
    hipMalloc((void**)&d_M, size*sizeof(float);
    hipMalloc((void**)&d_N, size*sizeof(float);
    hipMalloc((void**)&d_P, size*sizeof(float);

    







    //fill host matrices
    for (int i = 0; i < num_row; i++) {
        for (int j = 0; j < num_col; j++) {
            h_M[i][j] = ((float)rand() / RAND_MAX) * 100.0; //fill with rand values from 0-100
            h_N[i][j] = ((float)rand() / RAND_MAX) * 100.0;
        }
    }

    //Host matrix multiplication
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            h_P[i][j] = 0;
            for (int k = 0; k < n; k++) {
                h_P[i][j] += h_M[i][k] * h_N[k][j];
            }
        }
    }

    //fill device matrices

    //Cpy to dev, timer

    //Device Matrix multiplication
    mulKernel << <dimBlock, dimGrid, 0, 0 >> > (d_M, d_N, d_P, size);
    
    //Cpy to dev, timer

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
}
